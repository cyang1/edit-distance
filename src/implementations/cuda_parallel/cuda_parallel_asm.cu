#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define WARP_SIZE 32
// Assumed to be power of 2 for optimizations.
#define QUAD_SIZE 64
#define MAX_INPUT_LENGTH (WARP_SIZE * QUAD_SIZE)

#define MAX_ULL 0xFFFFFFFFFFFFFFFFULL
#define ULL_MSB (MAX_ULL ^ (MAX_ULL >> 1))

__device__ unsigned int device_min_idx, device_min_distance;

__device__ void
arbitrary_precision_add(const uint64_t *addend,
    const uint64_t *augend,
    uint32_t num_parts,
    uint64_t *result)
{
    const uint32_t *addend_32 = reinterpret_cast<const uint32_t*>(addend);
    const uint32_t *augend_32 = reinterpret_cast<const uint32_t*>(augend);
    uint32_t *result_32 = reinterpret_cast<uint32_t*>(result);

    asm("add.cc.u32 %0, %1, %2;\n"
        : "=r"(result_32[0])
        : "r"(addend_32[0]), "r"(augend_32[0]));
    for (int i = 0; i < 2 * (num_parts - 1); i++)
        asm("addc.cc.u32 %0, %1, %2;\n"
            : "=r"(result_32[i])
            : "r"(addend_32[i]), "r"(augend_32[i]));
    asm("addc.u32 %0, %1, %2;\n"
        : "=r"(result_32[2 * num_parts - 1])
        : "r"(addend_32[2 * num_parts - 1]), "r"(augend_32[2 * num_parts - 1]));
}

// Follows the method from http://www.gersteinlab.org/courses/452/09-spring/pdf/Myers.pdf
__global__ void
edit_distance_warp_kernel(const unsigned char *body,
    const unsigned int body_len,
    const unsigned char *input,
    const unsigned int input_len)
{
    unsigned int index = threadIdx.x;

    // Throw out threads that we don't need.
    int num_threads_necessary = (input_len + QUAD_SIZE - 1) / QUAD_SIZE;

    if (index >= num_threads_necessary)
        return;

    bool is_last_thread = index == (num_threads_necessary - 1);

    // Precompute each thread's portion of Peq[\sigma]
    uint64_t char_eq[UCHAR_MAX];
    for (int i = 0; i < UCHAR_MAX; i++) {
        char_eq[i] = 0;
    }
    __syncthreads();
    for (int i = index * QUAD_SIZE; i < min(input_len, (index + 1) * QUAD_SIZE); i++) {
        // Equivalent to (1 << (i % QUAD_SIZE))
        char_eq[input[i]] |= (1ull << (i & (QUAD_SIZE - 1)));
    }

    // Only the horizontal deltas need to be shared.
    uint64_t Mv = 0;                                        // Mv = 0
    __shared__ uint64_t Pv[WARP_SIZE], Ph[WARP_SIZE], Mh[WARP_SIZE], Eq_Pv[WARP_SIZE], Xh[WARP_SIZE];
    __shared__ unsigned long long carry, carry_through;

    Pv[index] = MAX_ULL;

    // Only matters for the last thread in the block.
    uint32_t score = input_len;                             // Score = m
    uint32_t min_score = input_len;
    uint32_t min_idx = 0;

    for (int j = 0; j < body_len; j++) {
        uint64_t Eq = char_eq[body[j]];
        uint64_t Xv = Eq | Mv;

        if (threadIdx.x == 0) {
            Eq_Pv[index] = Eq & Pv[index];
            arbitrary_precision_add(Eq_Pv, Pv, num_threads_necessary, Xh);
        }
        Xh[index] = (Xh[index] ^ Pv[index]) | Eq;
        // Computing Xh is hard because of carrying in the addition.
        // if (threadIdx.x == 0) {
        //     carry = 0;
        //     carry_through = 0;
        // }
        // uint64_t Eq_Pv = Eq & Pv;
        // uint64_t add_result = Eq_Pv + Pv;
        // // Should carry
        // if (add_result < Eq_Pv) atomicAdd(&carry, 2ull << index);
        // if (add_result == MAX_ULL) atomicAdd(&carry_through, 1ull << index);
        // if (threadIdx.x == 0) {
        //     // This is a problem...recursive?
        //     carry |= (((carry & carry_through) + carry_through) ^ carry_through);
        // }
        // if (carry & (1ull << index)) add_result++;
        // Xh[index] = (add_result ^ Pv) | Eq;
        // Xh[index] = (((Eq & Pv) + Pv) ^ Pv) | Eq;

        Ph[index] = Mv | ~(Xh[index] | Pv[index]);
        Mh[index] = Pv[index] & Xh[index];

        if (is_last_thread) {
            // Equivalent to (1 << ((input_len - 1) % QUAD_SIZE))
            const uint64_t last_bit = 1ull << ((input_len - 1) & (QUAD_SIZE - 1));
            if (Ph[index] & last_bit)
                score++;
            else if (Mh[index] & last_bit)
                score--;

            if (score < min_score) {
                min_score = score;
                min_idx = j;
            }
        }

        // Have to ensure memory storage to Ph and Mh is done before
        // this next step.
        __syncthreads();

        uint8_t last_bit_p = 0, last_bit_m = 0;
        if (index != 0) {
            last_bit_p = (Ph[index - 1] & ULL_MSB) != 0;
            last_bit_m = (Mh[index - 1] & ULL_MSB) != 0;
        }

        Ph[index] = (Ph[index] << 1) | last_bit_p;
        Mh[index] = (Mh[index] << 1) | last_bit_m;

        Pv[index] = Mh[index] | ~(Xv | Ph[index]);
        Mv = Ph[index] & Xv;
    }

    if (is_last_thread) {
        device_min_idx = min_idx;
        device_min_distance = min_score;
    }
}

void
edit_distance_cuda(std::string s1, std::string s2) {

    // compute number of blocks and threads per block
    const int threadsPerBlock = WARP_SIZE;
    const int blocks = 1;

    // s2 must be shorter
    if (s2.length() > s1.length()) {
        std::string tempStr = s1;
        s1 = s2;
        s2 = tempStr;
    }

    int n1 = s1.length();
    int n2 = s2.length();

    if (n2 > MAX_INPUT_LENGTH) {
        printf("This simple CUDA implementation can only support inputs "
               "of length up to 2048.\n");
        return;
    }

    unsigned char* device_s1;
    unsigned char* device_s2;

    hipMalloc(&device_s1, sizeof(unsigned char) * n1);
    hipMalloc(&device_s2, sizeof(unsigned char) * n2);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    // copy the input data to the graphics card
    hipMemcpy(device_s1, s1.c_str(), sizeof(unsigned char) * n1, hipMemcpyHostToDevice);
    hipMemcpy(device_s2, s2.c_str(), sizeof(unsigned char) * n2, hipMemcpyHostToDevice);

    // second timer that only checks computation time
    double startComputeTime = CycleTimer::currentSeconds();

    // run kernel
    edit_distance_warp_kernel<<<blocks, threadsPerBlock>>>(device_s1, n1, device_s2, n2);
    hipDeviceSynchronize();

    // get the duration of the computation
    double computeDuration = CycleTimer::currentSeconds() - startComputeTime;

    // copy the results back to the host
    unsigned int min_idx, min_distance;
    hipMemcpyFromSymbol(&min_idx, HIP_SYMBOL(device_min_idx), sizeof(min_idx), 0);
    hipMemcpyFromSymbol(&min_distance, HIP_SYMBOL(device_min_distance), sizeof(min_distance), 0);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Location: %d\n", min_idx);
    printf("Minimum Distance: %d\n", min_distance);
    printf("Overall: %.3f ms\n", 1000.f * overallDuration);
    printf("Actual Computation Time: %.3f ms\n", 1000.f * computeDuration);

    // free memory buffers on the GPU
    hipFree(device_s1);
    hipFree(device_s2);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
