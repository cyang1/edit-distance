#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define WARP_SIZE 32
// Assumed to be power of 2 for optimizations.
#define QUAD_SIZE 64
#define MAX_INPUT_LENGTH (QUAD_SIZE * QUAD_SIZE)
#define MAX_THREADS (MAX_INPUT_LENGTH / QUAD_SIZE)

#define MAX_ULL 0xFFFFFFFFFFFFFFFFULL
#define ULL_MSB (MAX_ULL ^ (MAX_ULL >> 1))

__device__ unsigned int device_min_idx, device_min_distance;

// Follows the method from http://www.gersteinlab.org/courses/452/09-spring/pdf/Myers.pdf
__global__ void
edit_distance_warp_kernel(const unsigned char* body,
    const unsigned int body_len,
    const unsigned char* input,
    const unsigned int input_len)
{
    unsigned int index = threadIdx.x;

    // Throw out threads that we don't need.
    int num_threads_necessary = (input_len + QUAD_SIZE - 1) / QUAD_SIZE;

    if (index >= num_threads_necessary)
        return;

    bool is_last_thread = index == (num_threads_necessary - 1);

    // Precompute each thread's portion of Peq[\sigma]
    uint64_t char_eq[UCHAR_MAX];
    for (int i = 0; i < UCHAR_MAX; i++) {
        char_eq[i] = 0;
    }
    __syncthreads();
    for (int i = index * QUAD_SIZE; i < min(input_len, (index + 1) * QUAD_SIZE); i++) {
        // Equivalent to (1 << (i % QUAD_SIZE))
        char_eq[input[i]] |= (1ull << (i & (QUAD_SIZE - 1)));
    }

    // Only the horizontal deltas need to be shared.
    uint64_t Pv = MAX_ULL,                                  // Pv = 1^m
             Mv = 0;                                        // Mv = 0
    __shared__ uint64_t Ph[MAX_THREADS], Mh[MAX_THREADS];
    __shared__ unsigned long long carry, carry_through;

    // Only matters for the last thread in the block.
    uint32_t score = input_len;                             // Score = m
    uint32_t min_score = input_len;
    uint32_t min_idx = 0;

    for (int j = 0; j < body_len; j++) {
        uint64_t Eq = char_eq[body[j]];
        uint64_t Xv = Eq | Mv;

        uint64_t Xh;
        if (num_threads_necessary == 1) {
            Xh = (((Eq & Pv) + Pv) ^ Pv) | Eq;
        } else {
            // Computing Xh is hard because of carrying in the addition.
            if (threadIdx.x == 0) {
                carry = 0;
                carry_through = 0;
            }
            uint64_t Eq_Pv = Eq & Pv;
            uint64_t add_result = Eq_Pv + Pv;
            // Should carry
            if (add_result < Eq_Pv) atomicAdd(&carry, 2ull << index);
            if (add_result == MAX_ULL) atomicAdd(&carry_through, 1ull << index);
            if (threadIdx.x == 0) {
                // This is a problem...recursive?
                carry |= (((carry & carry_through) + carry_through) ^ carry_through);
            }
            if (carry & (1ull << index)) add_result++;
            Xh = (add_result ^ Pv) | Eq;
        }

        Ph[index] = Mv | ~(Xh | Pv);
        Mh[index] = Pv & Xh;

        if (is_last_thread) {
            // Equivalent to (1 << ((input_len - 1) % QUAD_SIZE))
            const uint64_t last_bit = 1ull << ((input_len - 1) & (QUAD_SIZE - 1));
            if (Ph[index] & last_bit)
                score++;
            else if (Mh[index] & last_bit)
                score--;

            if (score < min_score) {
                min_score = score;
                min_idx = j;
            }
        }

        // Have to ensure memory storage to Ph and Mh is done before
        // this next step.
        __syncthreads();

        uint8_t last_bit_p = 0, last_bit_m = 0;
        if (index != 0) {
            last_bit_p = (Ph[index - 1] & ULL_MSB) != 0;
            last_bit_m = (Mh[index - 1] & ULL_MSB) != 0;
        }

        Ph[index] = (Ph[index] << 1) | last_bit_p;
        Mh[index] = (Mh[index] << 1) | last_bit_m;

        Pv = Mh[index] | ~(Xv | Ph[index]);
        Mv = Ph[index] & Xv;
    }

    if (is_last_thread) {
        device_min_idx = min_idx;
        device_min_distance = min_score;
    }
}

void
edit_distance_cuda(const char *body,
    const unsigned int body_len,
    const char *input,
    const unsigned int input_len,
    unsigned int& min_idx,
    unsigned int& min_distance)
{
    // compute number of blocks and threads per block
    const int blocks = 1, threadsPerBlock = (input_len + QUAD_SIZE - 1) / QUAD_SIZE;

    if (input_len > MAX_INPUT_LENGTH) {
        printf("This simple CUDA implementation can only support inputs "
               "of length up to 2048.\n");
        return;
    }

    unsigned char *device_body, *device_input;

    hipMalloc(&device_body, sizeof(unsigned char) * body_len);
    hipMalloc(&device_input, sizeof(unsigned char) * input_len);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    // copy the input data to the graphics card
    hipMemcpy(device_body, body, sizeof(unsigned char) * body_len, hipMemcpyHostToDevice);
    hipMemcpy(device_input, input, sizeof(unsigned char) * input_len, hipMemcpyHostToDevice);

    // second timer that only checks computation time
    double startComputeTime = CycleTimer::currentSeconds();

    // run kernel
    edit_distance_warp_kernel<<<blocks, threadsPerBlock>>>(
        device_body, body_len, device_input, input_len);
    hipDeviceSynchronize();

    // get the duration of the computation
    double computeDuration = CycleTimer::currentSeconds() - startComputeTime;

    // copy the results back to the host
    hipMemcpyFromSymbol(&min_idx, HIP_SYMBOL(device_min_idx), sizeof(min_idx), 0);
    hipMemcpyFromSymbol(&min_distance, HIP_SYMBOL(device_min_distance), sizeof(min_distance), 0);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Location: %d\n", min_idx);
    printf("Minimum Distance: %d\n", min_distance);
    printf("Overall: %.3f ms\n", 1000.f * overallDuration);
    printf("Actual Computation Time: %.3f ms\n", 1000.f * computeDuration);

    // free memory buffers on the GPU
    hipFree(device_body);
    hipFree(device_input);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
